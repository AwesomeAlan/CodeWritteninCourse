#include <algorithm>
#include <iostream>
#include <numeric>
#include <random>
#include <vector>

#ifdef PARALLEL
#include <omp.h>
#endif

#include <hip/hip_runtime.h>

constexpr auto MaxNum{1000000};
constexpr auto NumClass{1 << 4};
constexpr auto Interval{MaxNum / NumClass};

template <typename T> auto populateRandomVector(std::vector<T> &Vec) {
  std::random_device RndDevice;
  std::mt19937 MersenneEngine{RndDevice()};
  std::uniform_int_distribution<T> Dist{0, MaxNum};
  std::generate(Vec.begin(), Vec.end(),
                [&Dist, &MersenneEngine]() { return (Dist(MersenneEngine)); });
}

template <typename T> auto histogramCpu(const std::vector<T> &Vec) {
#ifdef PARALLEL
#pragma omp parallel for
#endif
  std::vector<int> Hist(NumClass);
  for (auto I{0}; I < Vec.size(); ++I) {
    Hist[Vec[I] / Interval]++;
  }
  for (int i = 0; i < NumClass; i++) {
	   std::cout << Hist[i] << " ";
  }
  std::cout << std::endl;
  return Hist;
}

template <typename T>
__global__ void histogramGpu(const T *Vec, const std::size_t N,
                             const int Interval, int *Hist) {
  auto Tid{blockIdx.x * blockDim.x + threadIdx.x};
  if (Tid >= N)
    return;
  atomicAdd(&Hist[Vec[Tid] / Interval], 1);
}

int main(int argc, char **argv) {
  using T = int;

  //auto N{1 << 28};
  for (int i = 10; i <= 25; i += 5) {
  auto N = 1 << i;
  auto NumberSize{N * sizeof(T)};
  auto HistSize{NumClass * sizeof(int)};
  std::vector<T> HostVec(N);
  populateRandomVector(HostVec);

  T *DeviceVec = NULL;
  T *DeviceHist = NULL;

  float GpuElapsedTimeMs[4] = {0.0f, 0.0f, 0.0f, 0.0f};

  hipEvent_t Start, Stop;
  hipEventCreate(&Start);
  hipEventCreate(&Stop);

  // Allocate memory
  hipEventRecord(Start, 0);
  hipMalloc(reinterpret_cast<void **>(&DeviceVec), NumberSize);
  hipMalloc(reinterpret_cast<void **>(&DeviceHist), HistSize);
  hipMemset(DeviceHist, 0, HistSize);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[0], Start, Stop);

  // Copy data to device
  hipEventRecord(Start, 0);
  hipMemcpy(DeviceVec, HostVec.data(), NumberSize, hipMemcpyHostToDevice);
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[1], Start, Stop);

  // Kernel configuration
  hipEventRecord(Start, 0);
  dim3 ThreadsPerBlock (128);
  dim3 BlocksPerGrid = ((N + 127) / 128);

  histogramGpu<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceVec, N, Interval,
                                                   DeviceHist);
  hipDeviceSynchronize();

  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[2], Start, Stop);

  // copy it back
  hipEventRecord(Start, 0);
  std::vector<T> FromDeviceHist(NumClass);
  hipMemcpy(FromDeviceHist.data(), DeviceHist, HistSize,
                   hipMemcpyDeviceToHost);

  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&GpuElapsedTimeMs[3], Start, Stop);

  // CPU
  auto CpuElapsedTimeMs{0.0f};
  hipEventRecord(Start, 0);
  auto HostHist{histogramCpu(HostVec)};
  hipEventRecord(Stop, 0);
  hipEventSynchronize(Stop);
  hipEventElapsedTime(&CpuElapsedTimeMs, Start, Stop);
  
  std::cout << "N = " << N << std::endl;
  std::cout << "Check passed? " << std::boolalpha
            << (FromDeviceHist == HostHist) << std::endl;
  std::cout << "GPU Time [ms]: ";
  std::for_each(std::begin(GpuElapsedTimeMs), std::end(GpuElapsedTimeMs),
                [&](float I) { std::cout << I << " "; });
  std::cout << std::endl;
  std::cout << "CPU Time [ms]: " << CpuElapsedTimeMs << std::endl;
  std::cout << "Speed up [xN]: "
            << CpuElapsedTimeMs /
                   (std::accumulate(std::begin(GpuElapsedTimeMs),
                                    std::end(GpuElapsedTimeMs), 0.0f))
            << " " << CpuElapsedTimeMs / GpuElapsedTimeMs[2] << std::endl;
  std::cout << "**********************************" << std::endl;
  hipFree(DeviceVec);
  hipFree(DeviceHist);
  }
  return EXIT_SUCCESS;
}

