#include <algorithm>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <iomanip>
#include <numeric>

#ifdef PARALLEL
#include <omp.h>
#endif

#include <hip/hip_runtime.h>

constexpr auto NThread{1024};
constexpr auto NIter{1000000};

__global__ void setupRandomState(hiprandState *State) {
  auto Tid{threadIdx.x + blockDim.x * blockIdx.x};
  if (Tid > NThread)
    return;
  hiprand_init(1234, Tid, 0, &State[Tid]);
}

__device__ void calculatePi(hiprandState *State, unsigned int Tid, int &Count) {
  auto DevCount{0};
  for (auto I{0}; I < NIter; ++I) {
    auto X{hiprand_uniform(&State[Tid])};
    auto Y{hiprand_uniform(&State[Tid])};
    DevCount += (std::pow(X, 2) + std::pow(Y, 2) <= 1);
  }
  Count = DevCount;
}

__global__ void calculatePiGpu(int *Sum, hiprandState *State) {
  auto Tid{blockIdx.x * blockDim.x + threadIdx.x};
  if (Tid > NThread)
    return;
  auto DevCount {0};
  calculatePi(State, Tid, DevCount);
  atomicAdd(&Sum[Tid], DevCount);
}

int main(int argc, char **argv) {
  using T = double;
  auto TotSize{sizeof(int) * NThread};

  int ThreadsPerBlock = 128;
  int BlocksPerGrid = (NThread + ThreadsPerBlock - 1) / ThreadsPerBlock;

  // Set random state
  hiprandState *State;
  hipMalloc(&State, sizeof(hiprandState) * NThread);
  setupRandomState<<<BlocksPerGrid, ThreadsPerBlock>>>(State);

  int *HostSum = nullptr;
  int *DeviceSum = nullptr;
  HostSum = (int *)malloc(TotSize);
  memset(HostSum, 0, TotSize);
  hipMalloc(reinterpret_cast<void **>(&DeviceSum), TotSize);
  hipMemset(DeviceSum, 0, TotSize);

  calculatePiGpu<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceSum, State);
  hipMemcpy(HostSum, DeviceSum, TotSize, hipMemcpyDeviceToHost);

    // std::for_each(HostSum, HostSum + NThread,
    //               [&](T I) { std::cout << I << " "; });
  std::cout << std::setprecision(32) << static_cast<T> (std::accumulate(HostSum, HostSum+NThread, 0) * 4.0 / NThread / NIter) << std::endl;

  return EXIT_SUCCESS;
}

